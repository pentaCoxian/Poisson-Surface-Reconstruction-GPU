/*****************************************************************//**
 * \file   CommonUtils.cu
 * \brief  文件主要包含一些常用的cuda工具函数
 * 
 * \author LUO
 * \date   January 12th 2024
 *********************************************************************/
#include "CommonUtils.h"


hipCtx_t SparseSurfelFusion::initCudaContext(int selected_device) {
    //初始化Cuda驱动的API
    CHECKCUDADRIVER(hipInit(0));

    //Query the device
    int device_count = 0;
    CHECKCUDADRIVER(hipGetDeviceCount(&device_count));
    for (auto dev_idx = 0; dev_idx < device_count; dev_idx++) {
        char dev_name[256] = { 0 };
        CHECKCUDADRIVER(hipDeviceGetName(dev_name, 256, dev_idx));
        printf("device %d: %s\n", dev_idx, dev_name);
    }

    //选择GPU
    printf("设备 %d 被用作并行处理器.\n", selected_device);
    hipDevice_t cuda_device;
    CHECKCUDADRIVER(hipDeviceGet(&cuda_device, selected_device));

    //创建cuda上下文
    hipCtx_t cuda_context;
    CHECKCUDADRIVER(hipCtxCreate(&cuda_context, hipDeviceScheduleAuto, cuda_device));
    return cuda_context;
}

void SparseSurfelFusion::destroyCudaContext(hipCtx_t context)
{
    hipDeviceSynchronize();
    CHECKCUDADRIVER(hipCtxDestroy(context));
}



void SparseSurfelFusion::createDefault2DTextureDescriptor(hipTextureDesc& descriptor)
{
    memset(&descriptor, 0, sizeof(descriptor));
    // 纹理寻址模式，使用3个维度(实际上只使用2维)
    descriptor.addressMode[0] = hipAddressModeBorder;  // 在边界之外返回0
    descriptor.addressMode[1] = hipAddressModeBorder;
    descriptor.addressMode[2] = hipAddressModeBorder;
    // 从纹理获取时要使用的过滤模式
    descriptor.filterMode = hipFilterModePoint;        // 最邻近插值--hipFilterModePoint       双线性插值--hipFilterModeLinear
    // 指定是否应将整数数据转换为浮点数
    descriptor.readMode = hipReadModeElementType;      // 读数据以指定的数据类型读，不全部转化成float
    // 是否将纹理坐标标准化
    descriptor.normalizedCoords = 0;                    // 不使用归一化纹理内存

}

void SparseSurfelFusion::createDefault2DResourceDescriptor(hipResourceDesc& descriptor, hipArray_t& hipArray)
{
    memset(&descriptor, 0, sizeof(hipResourceDesc));   // 资源描述子初值为0
    // 使用CUDA数组--hipResourceTypeArray      
    // 使用CUDA映射数组--hipResourceTypeMipmappedArray      
    // 使用设备上一段线性内存--hipResourceTypeLinear
    // 使用设备上一个2D块资源
    descriptor.resType = hipResourceTypeArray;         
    descriptor.res.array.array = hipArray;             // 将值的内存段赋入
}

void SparseSurfelFusion::createDepthTexture(const unsigned int rows, const unsigned int cols, hipTextureObject_t& texture, hipArray_t& hipArray)
{
    // 声明纹理描述
    hipTextureDesc depth_texture_desc;
    createDefault2DTextureDescriptor(depth_texture_desc);
    // 声明通道描述(只有一个通道的数据，数据类型是uint16)
    hipChannelFormatDesc depth_channel_desc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned); // 16bit的无符号整型
    // 分配cuda数组
    CHECKCUDA(hipMallocArray(&hipArray, &depth_channel_desc, cols, rows));
    // 声明资源描述
    hipResourceDesc resource_desc;
    createDefault2DResourceDescriptor(resource_desc, hipArray); // 初始化资源描述子并将资源数据hipArray赋值进去
    // 分配纹理内存
    CHECKCUDA(hipCreateTextureObject(&texture, &resource_desc, &depth_texture_desc, 0));
}

void SparseSurfelFusion::createDepthTextureSurface(const unsigned int rows, const unsigned int cols, hipTextureObject_t& texture, hipSurfaceObject_t& surface, hipArray_t& hipArray)
{
    //纹理描述
    hipTextureDesc depth_texture_description;
    createDefault2DTextureDescriptor(depth_texture_description);
    //创建通道描述
    hipChannelFormatDesc depth_channel_desc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
    //分配cuda数组
    CHECKCUDA(hipMallocArray(&hipArray, &depth_channel_desc, cols, rows));
    //创建资源desc
    hipResourceDesc resource_desc;
    createDefault2DResourceDescriptor(resource_desc, hipArray); // 初始化资源描述子并将资源数据hipArray赋值进去
    //分配纹理
    CHECKCUDA(hipCreateTextureObject(&texture, &resource_desc, &depth_texture_description, 0));
    CHECKCUDA(hipCreateSurfaceObject(&surface, &resource_desc));
}

void SparseSurfelFusion::createDepthTextureSurface(const unsigned int rows, const unsigned int cols, CudaTextureSurface& collect)
{
    createDepthTextureSurface(rows, cols,collect.texture, collect.surface, collect.hipArray);
}

void SparseSurfelFusion::createFloat1TextureSurface(const unsigned int rows, const unsigned int cols, hipTextureObject_t& texture, hipSurfaceObject_t& surface, hipArray_t& hipArray)
{
    //纹理描述
    hipTextureDesc float1_texture_desc;
    createDefault2DTextureDescriptor(float1_texture_desc);
    //创建通道描述，使用指定类型返回通道描述子，并填入每一个通道分量的bit数 (下述为1个通道，此通道数据位数为32bit)
    hipChannelFormatDesc float1_channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    //分配cuda数组
    CHECKCUDA(hipMallocArray(&hipArray, &float1_channel_desc, cols, rows));
    //创建资源desc
    hipResourceDesc resource_desc;
    createDefault2DResourceDescriptor(resource_desc, hipArray); // 初始化资源描述子并将资源数据hipArray赋值进去
    //分配纹理
    CHECKCUDA(hipCreateTextureObject(&texture, &resource_desc, &float1_texture_desc, 0));
    CHECKCUDA(hipCreateSurfaceObject(&surface, &resource_desc));
}

void SparseSurfelFusion::createFloat1TextureSurface(const unsigned int rows, const unsigned int cols, CudaTextureSurface& textureCollect)
{
    createFloat1TextureSurface(rows, cols, textureCollect.texture, textureCollect.surface, textureCollect.hipArray);
}

void SparseSurfelFusion::createFloat2TextureSurface(const unsigned int rows, const unsigned int cols, hipTextureObject_t& texture, hipSurfaceObject_t& surface, hipArray_t& hipArray)
{
    //纹理描述
    hipTextureDesc float2_texture_desc;
    createDefault2DTextureDescriptor(float2_texture_desc);
    //创建通道描述，使用指定类型返回通道描述子，并填入每一个通道分量的bit数 (下述为2个通道，此通道数据位数为32bit)
    hipChannelFormatDesc float2_channel_desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
    //分配cuda数组
    CHECKCUDA(hipMallocArray(&hipArray, &float2_channel_desc, cols, rows));
    //创建资源desc
    hipResourceDesc resource_desc;
    createDefault2DResourceDescriptor(resource_desc, hipArray); // 初始化资源描述子并将资源数据hipArray赋值进去
    //分配纹理
    CHECKCUDA(hipCreateTextureObject(&texture, &resource_desc, &float2_texture_desc, 0));
    CHECKCUDA(hipCreateSurfaceObject(&surface, &resource_desc));
}

void SparseSurfelFusion::createFloat2TextureSurface(const unsigned int rows, const unsigned int cols, CudaTextureSurface& textureCollect)
{
    createFloat2TextureSurface(rows, cols, textureCollect.texture, textureCollect.surface, textureCollect.hipArray);
}

void SparseSurfelFusion::createUChar1TextureSurface(const unsigned rows, const unsigned cols, hipTextureObject_t& texture, hipSurfaceObject_t& surface, hipArray_t& hipArray)
{
    //纹理描述
    hipTextureDesc uchar1_texture_desc;
    createDefault2DTextureDescriptor(uchar1_texture_desc);
    //创建通道描述，使用指定类型返回通道描述子，并填入每一个通道分量的bit数 (下述为1个通道，此通道数据位数为8bit)
    hipChannelFormatDesc uchar1_channel_desc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    //分配cuda数组
    CHECKCUDA(hipMallocArray(&hipArray, &uchar1_channel_desc, cols, rows));
    //创建资源desc
    hipResourceDesc resource_desc;
    createDefault2DResourceDescriptor(resource_desc, hipArray); // 初始化资源描述子并将资源数据hipArray赋值进去
    //分配纹理
    CHECKCUDA(hipCreateTextureObject(&texture, &resource_desc, &uchar1_texture_desc, 0));
    CHECKCUDA(hipCreateSurfaceObject(&surface, &resource_desc));
}

void SparseSurfelFusion::createUChar1TextureSurface(const unsigned rows, const unsigned cols, CudaTextureSurface& textureCollect)
{
    createUChar1TextureSurface(rows, cols, textureCollect.texture, textureCollect.surface, textureCollect.hipArray);
}


void SparseSurfelFusion::createFloat4TextureSurface(const unsigned int rows, const unsigned int cols, hipTextureObject_t& texture, hipSurfaceObject_t& surface, hipArray_t& hipArray)
{
    // 声明并初始化纹理描述子
    hipTextureDesc float4_texture_desc; // 纹理描述子
    createDefault2DTextureDescriptor(float4_texture_desc); // 将纹理描述子初始化

    // 创造通道描述，使用指定类型返回通道描述子，并填入每一个通道分量的bit数
    // 使用float类型返回描述子，并且每个通道分量都是32bit位
    hipChannelFormatDesc float4_channel_desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat); // 返回了一个float类型的通道描述子

    // 根据通道描述子float4_channel_desc分配cuda的内存
    CHECKCUDA(hipMallocArray(&hipArray, &float4_channel_desc, cols, rows));

    // 创建资源描述子
    hipResourceDesc resource_desc;
    createDefault2DResourceDescriptor(resource_desc, hipArray); // 初始化资源描述子并将资源数据hipArray赋值进去

    // 分配纹理内存
    CHECKCUDA(hipCreateTextureObject(&texture, &resource_desc, &float4_texture_desc, 0));
    // 分配表面内存
    CHECKCUDA(hipCreateSurfaceObject(&surface, &resource_desc));
}

void SparseSurfelFusion::createFloat4TextureSurface(const unsigned int rows, const unsigned int cols, CudaTextureSurface& textureCollect)
{
    createFloat4TextureSurface(rows, cols, textureCollect.texture, textureCollect.surface, textureCollect.hipArray);
}

void SparseSurfelFusion::releaseTextureCollect(CudaTextureSurface& textureCollect)
{
    CHECKCUDA(hipDestroyTextureObject(textureCollect.texture));
    CHECKCUDA(hipDestroySurfaceObject(textureCollect.surface));
    CHECKCUDA(hipFreeArray(textureCollect.hipArray));
}

void SparseSurfelFusion::query2DTextureExtent(hipTextureObject_t texture, unsigned int& width, unsigned int& height)
{
    hipResourceDesc texture_res;
    cudaSafeCall(hipGetTextureObjectResourceDesc(&texture_res, texture));
    hipArray_t cu_array = texture_res.res.array.array;
    hipChannelFormatDesc channel_desc;
    hipExtent extent;
    unsigned int flag;
    cudaSafeCall(hipArrayGetInfo(&channel_desc, &extent, &flag, cu_array));
    width = extent.width;
    height = extent.height;
}
